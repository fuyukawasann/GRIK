// ssim_gpu.cpp
#include <hip/hip_runtime.h>

#include <math.h>

__global__ void ssim_kernel(const float* img1, const float* img2, float* out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;

        // Calculate mean and variance of image 1
        float mean1 = 0.0f, var1 = 0.0f;
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int nx = x + i, ny = y + j;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    float val = img1[ny * width + nx];
                    mean1 += val;
                    var1 += val * val;
                }
            }
        }
        mean1 /= 9.0f;
        var1 = var1 / 9.0f - mean1 * mean1;

        // Calculate mean and variance of image 2
        float mean2 = 0.0f, var2 = 0.0f;
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int nx = x + i, ny = y + j;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    float val = img2[ny * width + nx];
                    mean2 += val;
                    var2 += val * val;
                }
            }
        }
        mean2 /= 9.0f;
        var2 = var2 / 9.0f - mean2 * mean2;

        // SSIM
        float c1 = 0.01f, c2 = 0.03f;
        float ssim = (2 * mean1 * mean2 + c1) * (2 * sqrt(var1 * var2) + c2) /
                    ((mean1 * mean1 + mean2 * mean2 + c1) * (var1 + var2 + c2));
        out[idx] = ssim;
    }
}

extern "C" __global__ void ssim_gpu(const float* img1, const float* img2, float* out, int width, int height) {
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    ssim_kernel<<<grid_size, block_size>>>(img1, img2, out, width, height);
}
